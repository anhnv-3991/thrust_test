#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <error.h>
#include <sys/time.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include <string.h>
#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/fill.h>

int main(int argc, char **argv)
{
	int size = 1000;
	if (argc == 2) {
		size = atoi(argv[1]);
	}

	int *host_test = (int*)malloc(size * sizeof(int));
	int *dev_test;


	checkCudaErrors(hipMalloc(&dev_test, size * sizeof(int)));
	thrust::device_ptr<int> thrust_test = thrust::device_pointer_cast(dev_test);
	thrust::fill(thrust_test, thrust_test + size, 1);
	thrust::device_vector<int> dev_res(size);

	struct timeval start, end;
	gettimeofday(&start, NULL);
	thrust::exclusive_scan(thrust_test, thrust_test + size, dev_res.begin());
	gettimeofday(&end, NULL);
	thrust::copy(dev_res.begin(), dev_res.end(), host_test);

	thrust_test[1] = 0;
//	for (int i = 0; i < size; i++) {
//		printf("%d ", host_test[i]);
//	}

	printf("\n");
	printf("Elapsed time for prefix sum: %ld usecs\n", (end.tv_sec - start.tv_sec) * 1000000 + (end.tv_usec - start.tv_usec));

	return 0;
}
